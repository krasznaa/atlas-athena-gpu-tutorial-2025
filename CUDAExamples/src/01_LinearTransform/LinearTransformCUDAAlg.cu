#include "hip/hip_runtime.h"
// Copyright (C) 2002-2025 CERN for the benefit of the ATLAS collaboration

// Local include(s).
#include "LinearTransformCUDAAlg.h"

// CUDA include(s).
#include <hip/hip_runtime.h>

/// Helper macro for checking CUDA calls
#define ATH_CUDA_CHECK(EXP)                                               \
   do                                                                     \
   {                                                                      \
      const hipError_t ce = EXP;                                         \
      if (ce != hipSuccess)                                              \
      {                                                                   \
         ATH_MSG_ERROR("Failed to execute \"" << #EXP << "\" because:"    \
                                              << hipGetErrorString(ce)); \
         return StatusCode::FAILURE;                                      \
      }                                                                   \
   } while (false)

namespace GPUTutorial
{
   namespace Kernels
   {
      /// Simple kernel for a parallel linear transformation.
      __global__ void linearTransform(std::size_t n, const float *input,
                                      float *output)
      {
         // Stop right away if we're out of bounds.
         const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
         if (i >= n) {
            return;
         }

         // Perform a very simple linear transformation.
         output[i] = 2.0f * input[i] + 1.0f;
      }

   } // namespace Kernels

   StatusCode LinearTransformCUDAAlg::initialize()
   {
      // Simply greet the user.
      ATH_MSG_INFO("Initializing " << name() << "...");

      // Return gracefully.
      return StatusCode::SUCCESS;
   }

   StatusCode LinearTransformCUDAAlg::execute(const EventContext &ctx) const
   {
      // Set up an input array on the host.
      constexpr std::size_t n = 1000000;
      std::vector<float> inputHost(n);
      for (std::size_t i = 0; i < n; ++i)
      {
         inputHost[i] = static_cast<float>(i);
      }

      // Allocate input and output buffers on the device.
      float *inputDevice = nullptr;
      float *outputDevice = nullptr;
      ATH_CUDA_CHECK(hipMalloc(&inputDevice, n * sizeof(float)));
      ATH_CUDA_CHECK(hipMalloc(&outputDevice, n * sizeof(float)));

      // Copy the input data to the device.
      ATH_CUDA_CHECK(hipMemcpy(inputDevice, inputHost.data(),
                                n * sizeof(float), hipMemcpyHostToDevice));

      // Run the kernel.
      const std::size_t blockSize = 256;
      const std::size_t numBlocks = (10000000 + blockSize - 1) / blockSize;
      Kernels::linearTransform<<<numBlocks, blockSize>>>(n, inputDevice,
                                                         outputDevice);
      ATH_CUDA_CHECK(hipGetLastError());
      ATH_CUDA_CHECK(hipDeviceSynchronize());

      // Copy the output data back to the host.
      std::vector<float> outputHost(n);
      ATH_CUDA_CHECK(hipMemcpy(outputHost.data(), outputDevice,
                                n * sizeof(float), hipMemcpyDeviceToHost));

      // Print some elements of the output.
      ATH_MSG_INFO("outputHost[0]      = " << outputHost[0]);
      ATH_MSG_INFO("outputHost[1000]   = " << outputHost[1000]);
      ATH_MSG_INFO("outputHost[999999] = " << outputHost[999999]);

      // Return gracefully.
      return StatusCode::SUCCESS;
   }

} // namespace GPUTutorial
