#include "hip/hip_runtime.h"
// Copyright (C) 2002-2025 CERN for the benefit of the ATLAS collaboration

// Local include(s).
#include "JetPullCUDAAlg.h"

// Framework include(s).
#include "AthenaKernel/errorcheck.h"

/// Helper macro for checking CUDA calls
#define ATH_CUDA_CHECK(EXP)                                             \
   do                                                                   \
   {                                                                    \
      const hipError_t ce = EXP;                                       \
      if (ce != hipSuccess)                                            \
      {                                                                 \
         REPORT_ERROR_WITH_CONTEXT(StatusCode::FAILURE,                 \
                                   "GPUTutorial::JetPullCUDAAlg")       \
             << "Failed to execute \""                                  \
             << #EXP << "\" :"                                          \
             << hipGetErrorName(ce) << ": " << hipGetErrorString(ce); \
         return StatusCode::FAILURE;                                    \
      }                                                                 \
   } while (false)

namespace GPUTutorial
{
   namespace Kernels
   {
      /// Simple kernel "calibrating" electrons
      __global__ void
      calibrateElectrons(ElectronDeviceContainer::const_view inputView,
                         ElectronDeviceContainer::view outputView)
      {
         // Get the index of the current thread.
         const int idx = blockIdx.x * blockDim.x + threadIdx.x;

         // Construct the device containers.
         const ElectronDeviceContainer::const_device input(inputView);
         ElectronDeviceContainer::device output(outputView);

         // Check if the index is within bounds.
         if (idx >= input.size())
         {
            return;
         }

         // Copy the input electron to the output container.
         output[idx].eta() = input[idx].eta();
         output[idx].phi() = input[idx].phi();

         // Perform some calibration on the output electron.
      }

   } // namespace Kernels

   StatusCode JetPullCUDAAlg::deviceExecute(const std::span<const float>& jetPt, ///< [in] Jet pT array
                                            const std::span<const float>& jetEta, ///< [in] Jet eta array
                                            const std::span<const float>& jetPhi, ///< [in] jet phi array
                                            const std::pmr::vector<std::size_t>& nConstituents, ///< [in] number of constituents for each jet
                                            const std::pmr::vector<float>& constPt, ///< [in] flat array of constituent pTs (grouped by jet)
                                            const std::pmr::vector<float>& constEta, ///< [in] flat array of constituent etas (grouped by jet)
                                            const std::pmr::vector<float>& constPhi, ///< [in] flat array of constituent phis (grouped by jet)
                                            std::pmr::vector<float>& jetPullY, ///< [out] rapidity component of each jet pull vector
                                            std::pmr::vector<float>& jetPullPhi ///< [out] phi component of each jet pull vector
                                          ) const
   {
      
   }

} // namespace GPUTutorial
