#include "hip/hip_runtime.h"
// Copyright (C) 2002-2025 CERN for the benefit of the ATLAS collaboration

// Local include(s).
#include "calibrateElectrons.h"

// Framework include(s).
#include "AthenaKernel/errorcheck.h"

/// Helper macro for checking CUDA calls
#define ATH_CUDA_CHECK(EXP)                                           \
   do                                                                 \
   {                                                                  \
      const hipError_t ce = EXP;                                     \
      if (ce != hipSuccess)                                          \
      {                                                               \
         REPORT_ERROR_WITH_CONTEXT(StatusCode::FAILURE,               \
                                   "GPUTutorial::calibrateElectrons") \
             << "Failed to execute \""                                \
             << #EXP << "\" because:"                                 \
             << hipGetErrorString(ce);                               \
         return StatusCode::FAILURE;                                  \
      }                                                               \
   } while (false)

namespace GPUTutorial
{
   namespace Kernels
   {
      /// Simple kernel "calibrating" electrons
      __global__ void
      calibrateElectrons(ElectronDeviceContainer::const_view inputView,
                         ElectronDeviceContainer::view outputView)
      {
         // Get the index of the current thread.
         const int idx = blockIdx.x * blockDim.x + threadIdx.x;

         // Construct the device containers.
         const ElectronDeviceContainer::const_device input(inputView);
         ElectronDeviceContainer::device output(outputView);

         // Check if the index is within bounds.
         if (idx >= input.size())
         {
            return;
         }

         // Copy the input electron to the output container.
         output[idx].eta() = input[idx].eta();
         output[idx].phi() = input[idx].phi();

         // Perform some calibration on the output electron.
      }

   } // namespace Kernels

   StatusCode calibrateElectrons(ElectronDeviceContainer::const_view input,
                                 ElectronDeviceContainer::view output)
   {
      // Launch the kernel.
      const int blockSize = 256;
      const int numBlocks = (input.capacity() + blockSize - 1) / blockSize;
      Kernels::calibrateElectrons<<<numBlocks, blockSize>>>(input, output);

      // Check for errors in kernel launch.
      ATH_CUDA_CHECK(hipGetLastError());

      // Wait for the device to finish with the kernel.
      ATH_CUDA_CHECK(hipDeviceSynchronize());

      // Return gracefully.
      return StatusCode::SUCCESS;
   }

} // namespace GPUTutorial
